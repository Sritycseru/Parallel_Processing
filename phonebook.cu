

#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;
__global__ void kmp(char *P, char *T, int *index, int p_len,int t_len,int number_of_core, int number_of_contact)
{
    int idx = threadIdx.x;
    int start_index = idx * (number_of_contact / number_of_core);
    int end_index = (idx + 1) * (number_of_contact / number_of_core);
    int star_count = start_index;
    if (idx == number_of_core - 1)
    {
        end_index = t_len;
    }else{
    	end_index = index[end_index];
    }
    int fnd=0;
    for (int i = index[start_index]; i < end_index; i++)
    {
        if(T[i]=='*'){fnd=0; star_count++;}
        if(T[i]==P[fnd]) fnd++;
        
        else fnd=0;
        
        if(fnd==p_len){
            index[star_count]= -1;
            fnd=0;
        }
    }
}

int main(int argc ,char *argv[]){

    string name="Jodey";
    vector<string>contact_list;
    vector<string> file_names(argv + 1, argv + argc);
        for (auto file_name: file_names)
        {
            ifstream file(file_name);
            string text1;
            while (getline(file,text1))
            {
                contact_list.push_back(text1);
            }
            file.close();
        }


    int number_of_core = 4;
    

    string text = "";
   int number_of_contact=contact_list.size();
   // cout<<number_of_contact<<endl;

    vector <int> contact_list_index(number_of_contact);
   for(int i = 0 ; i < number_of_contact; i++){
        contact_list_index[i]=text.size();
        text += contact_list[i] + "*";
    }
    char *DT, *PT;
    int m = text.size() + 1;
    hipMalloc(&DT, m);
    hipMemcpy(DT, &text[0], m, hipMemcpyHostToDevice);

    int p_len = name.size();
    hipMalloc(&PT, p_len);
    hipMemcpy(PT, &name[0], p_len, hipMemcpyHostToDevice);

    int *index;

    hipMalloc(&index, number_of_contact * sizeof(int));
    hipMemcpy(index, &contact_list_index[0], number_of_contact * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start); //start time
    kmp<<<1, number_of_core>>>(PT,DT, index, p_len,m, number_of_core,number_of_contact);

    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    cout << "Time taken to the gpu: with " << number_of_core << " is : " << milliseconds << " miliseconds." << endl;
     vector<int> res(number_of_contact);
    hipMemcpy(&res[0], index, number_of_contact * sizeof(int), hipMemcpyDeviceToHost);
    set<string>st;

    for (int i = 1; i < number_of_contact; i++)
    {
        if (res[i] == -1)
            st.insert(contact_list[i]);

    }
    for(auto x:st){
      cout<<x<<endl;
    }
}