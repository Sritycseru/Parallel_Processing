
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

__global__ void matrix_multiplication(int *A, int *B, int *C, int N,int M,int P,int number_of_matrix,int number_of_core){



    int i = threadIdx.x;//rank
    int start_matrix_index = (number_of_matrix/ number_of_core) * i;
    int end_matrix_index = (number_of_matrix / number_of_core) * (i + 1);
    if(i == number_of_core - 1) end_matrix_index = number_of_matrix;

    for(int x = start_matrix_index; x < end_matrix_index; x++){

        for(int i = 0; i < N; i++){
            for(int j = 0; j < P; j++){
                int sum = 0;
                for(int k = 0; k < M; k++){
                    sum += A[ i * M + k] * B[k * P + j];
                }
                C[i * P + j] = sum;
            }
        }
    }
}
int main(int argc , char *argv[]){
    freopen("mul.txt","r",stdin);
    cout << "Enter the dimenssion of the matrix, N, M , P: " << endl;
    int N , M , P;
    cin >> N >> M >> P;
    cout << "Enter the number of Matrix: " << endl;
    int number_of_matrix;
    cin >> number_of_matrix;

    // totall size calculation
    int a_element = number_of_matrix * N * M;
    int b_element = number_of_matrix * M * P;
    int c_element = number_of_matrix * N * P;

    int *A=new int[a_element];
    int *B=new int[b_element];
    int *C=new int [c_element];

    for(int i = 0; i < a_element; i++){
        A[i]=1;
        //A[i] = (rand() % 3) + 1;
    }
    for(int i = 0; i < b_element; i++){
        //B[i] = (rand() % 3) + 1;
        B[i]=1;
    }


    int *DA, *DB, *DC;
    hipMalloc(&DA, a_element * sizeof(int));
    hipMalloc(&DB, b_element * sizeof(int));
    hipMalloc(&DC, c_element * sizeof(int)); //need after the multiplication
    //host holo cpu and device holo gpu
    hipMemcpy(DA, A, a_element * sizeof(int), hipMemcpyHostToDevice); //cpu theke value gpu te pass
    hipMemcpy(DB, B, b_element * sizeof(int) , hipMemcpyHostToDevice);

    cout << "Enter the number of core you want to use." << endl;
    int number_of_core=4;


    hipEvent_t start, end;
    
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);// time calculation of starting time

    matrix_multiplication <<< 1 , number_of_core >>> (DA, DB, DC, N , M , P,number_of_matrix, number_of_core);


    hipEventRecord(end);
    hipEventSynchronize(end);
    float time_totall = 0;
    hipEventElapsedTime(&time_totall, start, end);

    cout << "Time taken to the gpu: with " << number_of_core << " is : " << time_totall << " miliseconds." << endl;

    hipMemcpy(C, DC, c_element * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        for(int j = 0; j < P; j++){
            cout << C[i * P + j] << " ";
        }
        cout << endl;
    }

    return 0;

}